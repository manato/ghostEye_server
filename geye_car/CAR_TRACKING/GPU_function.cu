#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include"for_use_GPU.h"


/* declaration of texture memory */
//texture<FLOAT> A;
//texture<FLOAT> B;
texture<float, hipTextureType1D, hipReadModeElementType> A;
texture<float, hipTextureType1D, hipReadModeElementType> B;
texture<int2, hipTextureType1D, hipReadModeElementType> A_double;
texture<int2, hipTextureType1D, hipReadModeElementType> B_double;


//thread process
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// convolve A and B(non_symmetric)
//unsigned __stdcall process(void *thread_arg) {

/********************************************/
/* function for calculating root */
/********************************************/
extern "C"
__global__
void
process_root 
(
 //FLOAT *A,  
 //FLOAT *B, 
 FLOAT *C, 
 int *A_dims_array, 
 int *B_dims_array, 
 int len,
 int interval, 
 int L_MAX,
 int *error_array,
 int error_array_num,
 int pid,
 int device_number
) 
{
  int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  int ii = blockIdx.z % len;
  int level = blockIdx.z / len;

  int A_dims[3] = { A_dims_array[level*3], A_dims_array[level*3+1], A_dims_array[level*3+2] };
  int B_dims[3] = { B_dims_array[ii*3], B_dims_array[ii*3+1], B_dims_array[ii*3+2] };
  int C_dims[2] = { A_dims[0] - B_dims[0] + 1, A_dims[1] - B_dims[1] + 1 };

  int C_x = C_dims[1]/device_number;
  
  if(C_dims[1]%device_number != 0){
    C_x++;
  }
 
  idx_x = idx_x + pid * C_x;
 
  if(idx_x < C_x * pid  ||  idx_x >=  C_x * (pid + 1)){
    return ;
  }  

  if(0 <= ii && ii < len && 0 <= idx_x && idx_x < C_dims[1] && 0 <= idx_y && idx_y < C_dims[0] && interval <= level && level < L_MAX ) { 


    int num_features = A_dims[2];
    const int A_SQ = A_dims[0]*A_dims[1];
    const int B_SQ = B_dims[0]*B_dims[1];
    FLOAT add_val = 0;
    
    int x = idx_x;
    int y = idx_y;
    int XA0 = A_dims[0]*x;

    
    /* apply loop condition */
    for(int i=0; i<error_array_num; i++){
      if(error_array[i] == level){
        return;
      }
    }
    
    
    
    /* adjust the location of pointer of C */
    FLOAT *dst;
    unsigned long long int pointer = (unsigned long long int)C;

    for(int a=interval; a<level; a++) {
      for(int b=0; b<len; b++) {
        int height = A_dims_array[a*3] - B_dims_array[b*3] + 1; 
        int width = A_dims_array[a*3 + 1] - B_dims_array[b*3 + 1] + 1;
        
        /* error semantics */
        if (height < 1 || width < 1){
          printf("Invalid input in GPU\n");
          return;
        }
        
        pointer += (unsigned long long int)(height*width*sizeof(FLOAT));
       
      }
    }

    for(int b=0; b<ii; b++){
      int height = A_dims_array[level*3] - B_dims_array[b*3] + 1;
      int width  = A_dims_array[level*3 + 1] - B_dims_array[b*3 + 1] + 1;

      /* error semantics */
      if (height < 1 || width < 1){
        printf("Invalid input in GPU\n");
        return;
      }
      
      pointer += (unsigned long long int)(height*width*sizeof(FLOAT));
    }
    
    dst = (FLOAT *)pointer;
    
    /* adjust the location of pointer of A */
    //unsigned long long int pointerA = (unsigned long long int)A;
    int A_index_ini = 0;
    for(int a=0; a<level; a++) {
      //      pointerA += (unsigned long long int)(A_dims_array[a*3]*A_dims_array[a*3 + 1]*A_dims_array[a*3 + 2]*sizeof(FLOAT));
      A_index_ini += A_dims_array[a*3]*A_dims_array[a*3 + 1]*A_dims_array[a*3 + 2];
    }
    
    
    /* adjust the location of pointer of B */
    //unsigned long long int pointerB = (unsigned long long int)B;
    int B_index_ini = 0;
    for(int b=0; b<ii; b++) {
      //      pointerB += (unsigned long long int)(B_dims_array[b*3]*B_dims_array[b*3 + 1]*B_dims_array[b*3 + 2]*sizeof(FLOAT));
      B_index_ini += B_dims_array[b*3]*B_dims_array[b*3 + 1]*B_dims_array[b*3 + 2];
    } 

            
    for(int f = 0; f < num_features; f++) // num_features = 31
      {  
        // FLOAT *A_src = (FLOAT *)pointerA + f*A_SQ;      
        int A_index = A_index_ini + f*A_SQ;
        // FLOAT *B_src = (FLOAT *)pointerB + f*B_SQ;     
        int B_index = B_index_ini + f*B_SQ;
        
        // FLOAT *A_src2 =A_src+XA0; 
        A_index += XA0;

        FLOAT val = 0;
        // FLOAT *A_off = A_src2+y;
        A_index += y;
        // FLOAT *B_off = B_src;
        
        for (int xp = 0; xp < B_dims[1]; xp++) 
          {
            // FLOAT *A_temp = A_off;						
            int A_index_tmp = A_index;
            // FLOAT *B_temp = B_off;
            int B_index_tmp = B_index;
	  
            for (int yp = 0; yp < B_dims[0]; yp++) 	  
              {
                // val += *(A_temp++) * *(B_temp++);
                if(sizeof(FLOAT) == sizeof(float)) // if configured to use single precision
                  {
                    FLOAT A_val = tex1Dfetch(A, A_index_tmp);
                    FLOAT B_val = tex1Dfetch(B, B_index_tmp);
                    val += A_val * B_val;
                  } 
                else
                  {      // if configured to use double precision
                    int2 A_val = tex1Dfetch(A_double, A_index_tmp);
                    int2 B_val = tex1Dfetch(B_double, B_index_tmp);
                    val += __hiloint2double(A_val.y, A_val.x) * __hiloint2double(B_val.y, B_val.x);
                  }
                
                A_index_tmp++;
                B_index_tmp++;
              }
            
            // A_off+=A_dims[0];
            A_index += A_dims[0];
            // B_off+=B_dims[0];
            B_index += B_dims[0];
            
          }
        
        add_val += val;
      }
    
    *(dst + (idx_x*C_dims[0] + idx_y)) += add_val;
  }
  
  
  return;
}



/********************************************/
/* function for calculating part */
/********************************************/
extern "C"
__global__
void
process_part
(
 //FLOAT *A,  
 //FLOAT *B, 
 FLOAT *C, 
 int *A_dims_array, 
 int *B_dims_array, 
 int len,
 int interval, 
 int L_MAX,
 int *error_array,
 int error_array_num,
 int pid,
 int device_number
) 
{
  int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
  int ii = blockIdx.z % len;
  int level = blockIdx.z / len; 

  int A_dims[3] = { A_dims_array[level*3], A_dims_array[level*3+1], A_dims_array[level*3+2] };
  int B_dims[3] = { B_dims_array[ii*3], B_dims_array[ii*3+1], B_dims_array[ii*3+2] };
  int C_dims[2] = { A_dims[0] - B_dims[0] + 1, A_dims[1] - B_dims[1] + 1 };

  int C_x = C_dims[1]/device_number;

  if(C_dims[1]%device_number != 0){
    C_x++;
  }  
 
  idx_x = idx_x + pid * C_x;
 
  if(idx_x < C_x * pid  ||  idx_x >=  C_x * (pid + 1)){
    return ;
  }  

  if(0 <= ii && ii < len && 0 <= idx_x && idx_x < C_dims[1] && 0 <= idx_y && idx_y < C_dims[0] && 0 <= level && level < (L_MAX - interval) ) {
    int num_features = A_dims[2];
    const int A_SQ = A_dims[0]*A_dims[1];
    const int B_SQ = B_dims[0]*B_dims[1];
    FLOAT add_val = 0;

    int x = idx_x;
    int y = idx_y;
    int XA0 = A_dims[0]*x;
    
    /* apply loop condition */
    for(int i=0; i<error_array_num; i++){
      if(error_array[i] == level)
        return;
    }
    
    /* adjust the location of pointer of C */
    FLOAT *dst;
    unsigned long long int pointer = (unsigned long long int)C;
    for(int a=0; a<level; a++) {
      for(int b=0; b<len; b++){
        int height = A_dims_array[a*3] - B_dims_array[b*3] + 1;
        int width = A_dims_array[a*3 + 1] - B_dims_array[b*3 + 1] + 1;
        
        /* error semantics */
        if(height < 1 || width < 1){
          printf("Invalid input in GPU\n");
          return;
        }
        
        pointer += (unsigned long long int)(height*width*sizeof(FLOAT));
      }
    }

    for(int b=0; b<ii; b++){
      int height = A_dims_array[level*3] - B_dims_array[b*3] + 1;
      int width  = A_dims_array[level*3 + 1] - B_dims_array[b*3 + 1] + 1;

       /* error semantics */
        if(height < 1 || width < 1){
          printf("Invalid input in GPU\n");
          return;
        }

      pointer += (unsigned long long int)(height*width*sizeof(FLOAT));
    }
    

    dst = (FLOAT *)pointer;

    /* adjust the location of pointer of A */
    // unsigned long long int pointerA = (unsigned long long int)A;
    int A_index_ini = 0;
    for(int a=0; a<level; a++) {
      // pointerA += (unsigned long long int)(A_dims_array[a*3]*A_dims_array[a*3 + 1]*A_dims_array[a*3 + 2]*sizeof(FLOAT));
      A_index_ini += A_dims_array[a*3]*A_dims_array[a*3 + 1]*A_dims_array[a*3 + 2];
    }
    
    /* adjust the location of pointer of B */
    // unsigned long long int pointerB = (unsigned long long int)B;
    int B_index_ini = 0;
    for(int b=0; b<ii; b++) {
      // pointerB += (unsigned long long int)(B_dims_array[b*3]*B_dims_array[b*3 + 1]*B_dims_array[b*3 + 2]*sizeof(FLOAT));
      B_index_ini += B_dims_array[b*3]*B_dims_array[b*3 + 1]*B_dims_array[b*3 + 2];
    } 
    
    for(int f = 0; f < num_features; f++) // num_features = 31
      {  
        // FLOAT *A_src = (FLOAT *)pointerA + f*A_SQ;      
        int A_index = A_index_ini + f*A_SQ;
        // FLOAT *B_src = (FLOAT *)pointerB + f*B_SQ;     
        int B_index = B_index_ini + f*B_SQ;
        
        // FLOAT *A_src2 =A_src+XA0; 
        A_index += XA0;

        FLOAT val = 0;
        // FLOAT *A_off = A_src2+y;
        A_index += y;
        // FLOAT *B_off = B_src;
        
        for (int xp = 0; xp < B_dims[1]; xp++) 
          {
            // FLOAT *A_temp = A_off;						
            int A_index_tmp = A_index;
            // FLOAT *B_temp = B_off;	  
            int B_index_tmp = B_index;
 
            for (int yp = 0; yp < B_dims[0]; yp++) 	  
              {
                // val += *(A_temp++) * *(B_temp++);
                if(sizeof(FLOAT) == sizeof(float)) // if configured to use single precision
                  {
                    FLOAT A_val = tex1Dfetch(A, A_index_tmp);
                    FLOAT B_val = tex1Dfetch(B, B_index_tmp);
                    val += A_val * B_val;
                  }
                else            // if configured to use double precision
                  {
                    int2 A_val = tex1Dfetch(A_double, A_index_tmp);
                    int2 B_val = tex1Dfetch(B_double, B_index_tmp);
                    val += __hiloint2double(A_val.y, A_val.x) * __hiloint2double(B_val.y, B_val.x);
                  }
                
                A_index_tmp++;
                B_index_tmp++;
              }
            
            // A_off+=A_dims[0];
            A_index += A_dims[0];
            // B_off+=B_dims[0];
            B_index += B_dims[0];
            
          }
        add_val += val;
      }

    *(dst + (idx_x*C_dims[0] + idx_y)) += add_val;
  }
  
  return;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__
void
inverse_Q(
  FLOAT *src_start,
  int *size_array,
  int *error_array,
  int error_array_num,
  int NoP,
  int *PIDX_array,
  int *numpart,
  int NoC,
  int max_numpart,
  int interval,
  int L_MAX,
  int pid,
  int device_number
          )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int kk = blockIdx.y * blockDim.y + threadIdx.y;
  int jj = threadIdx.z;
  int L = blockIdx.z;
  int numpart_jj;
  int C_y;


  if(0<=jj && jj<NoC)
    {
      numpart_jj = numpart[jj];
      C_y = numpart_jj/device_number;
      if(numpart_jj%device_number != 0){
        C_y++;
       }
      kk = kk + pid * C_y;
      if(kk < C_y * pid  ||  kk >=  C_y * (pid + 1)){
         return ;
       }
    } else return ;
   

  if(0<=L && L < (L_MAX-interval)) 
    {
  
      /* loop condition */
      for(int h=0; h<error_array_num; h++) {
        if(L==error_array[h]){ 
          return;
        }
      }
    
     
      if( 0<=kk && kk < numpart_jj )
        {
          int PIDX = PIDX_array[L*(NoC*max_numpart) + jj*max_numpart + kk];
          int dim0 = size_array[L*NoP*2 + PIDX*2];
          int dim1 = size_array[L*NoP*2 + PIDX*2+1]; 

          if( idx < 0 || dim0*dim1 <= idx) return;

              /* pointer adjustment */
          FLOAT *src;
          unsigned long long int ptr_adjuster = (unsigned long long int)src_start;
          for(int i=0; i<L; i++) {
                
                /* apply error condition */
            int error_flag=0;
            for(int h=0; h<error_array_num; h++) {
              if(i==error_array[h]){
                error_flag = 1;
              }
            }
            if(error_flag != 0) {
              continue;
            }

                
            for(int j=0; j<NoP; j++) {
              int height = size_array[i*NoP*2 + j*2];
              int width = size_array[i*NoP*2 + j*2+1];
              ptr_adjuster += (unsigned long long int)(height*width*sizeof(FLOAT));
                  
            }
          }
              
   
              
          for(int j=0; j<PIDX; j++) {
            int height = size_array[L*NoP*2 + j*2];
            int width = size_array[L*NoP*2 + j*2+1];
            ptr_adjuster += (unsigned long long int)(height*width*sizeof(FLOAT));
          }
              
          src = (FLOAT *)ptr_adjuster;  
                        
          *(src + idx) *= -1;
        
      }
    }       
}



///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// dt helper function
__device__
void 
dt_helper(FLOAT *src, FLOAT *dst, int *ptr, int step, int s1, int s2, int d1, int d2, FLOAT a, FLOAT b) 
{
  if (d2 >= d1) 
    {
      int d = (d1+d2) >> 1;
      int ds =d*step;
      int s = s1;
      FLOAT src_ss = *(src+s*step);
      for (int p = s1+1; p <= s2; p++)
        {
          int t1 = d-s;
          int t2 = d-p;
          if (src_ss + a*t1*t1 + b*t1 > *(src+p*step) + a*t2*t2 + b*t2) 
            {
              s = p;
              src_ss = *(src+s*step);
            }
        }
      int D = d-s;
      dst[ds] = *(src+s*step) + a*D*D + b*D;
      ptr[ds] = s;
      dt_helper(src, dst, ptr, step, s1, s, d1, d-1, a, b);
      dt_helper(src, dst, ptr, step, s, s2, d+1, d2, a, b);
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//sub function of dt 
extern "C"
__global__
void 
dt1d_x(
  FLOAT *src_start,             // part_C_dev
  FLOAT *dst_start,             // tmpM_dev
  int *ptr_start,               // tmpIy_dev
  int *DID_4_array,             // DID_4_array_dev
  FLOAT *def_array,             // def_array_dev
  int *size_array,              // pm_size_array_dev
  int NoP,                      // NoP
  int *PIDX_array,              // PIDX_array_dev
  int *error_array,             // part_error_array_dev
  int error_array_num,          // part_error_array_num
  int *numpart,                 // numpart_jj
  int NoC,                      // NoC
  int max_numpart,              // max_numpart
  int interval,                 // interval
  int L_MAX,                     // L_MAX
  int pid,                       // pid
  int device_number              // device_number

       ) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int kk = blockIdx.y * blockDim.y + threadIdx.y;
  int jj = threadIdx.z;
  int L = blockIdx.z;
  int numpart_jj;
  int C_y;

  if(0<=jj && jj<NoC)
    {

      numpart_jj = numpart[jj];
      C_y = numpart_jj/device_number;

      if(numpart_jj%device_number != 0){
        C_y++;
       }
 
      kk = kk + pid * C_y;
 
      if(kk < C_y * pid  ||  kk >=  C_y * (pid + 1)){
         return ;
       }
    } else{
      return ;
    }


  if(0<=L && L<(L_MAX-interval)) 
    {
      /* loop condition */
      for(int h=0; h<error_array_num; h++) {
        if(L==error_array[h]){ 
          return;
        }
      }
                
      if(0<=kk && kk<numpart_jj)
        {
          int PIDX = PIDX_array[L*(NoC*max_numpart) + jj*max_numpart + kk];
          int dim1 = size_array[L*NoP*2 + PIDX*2+1]; 

          if( idx < 0 || dim1 <= idx ) return;

          int dim0 = size_array[L*NoP*2 + PIDX*2];
          int XD=0;
          int step = 1;
          int n = dim0;  
          int DID_4 = DID_4_array[L*(NoC*max_numpart) + jj*max_numpart + kk];
          FLOAT a = def_array[DID_4+2];
          FLOAT b = def_array[DID_4+3];
             
          /* pointer adjustment */
          unsigned long long int adj_src = (unsigned long long int)src_start;
          unsigned long long int adj_dst = (unsigned long long int)dst_start;
          unsigned long long int adj_ptr = (unsigned long long int)ptr_start;
          /* for src */
          for(int i=0; i<L; i++) {
                
            /* apply error condition */
            int error_flag=0;
            for(int h=0; h<error_array_num; h++) {
              if(i==error_array[h]){
                error_flag = 1;
              }
            }
            if(error_flag != 0) {
              continue;
            }
                
            for(int j=0; j<NoP; j++) {
              int height = size_array[i*NoP*2 + j*2];
              int width = size_array[i*NoP*2 + j*2+1];
              adj_src += (unsigned long long int)(height*width*sizeof(FLOAT));
                  
            }
          }
              
              
          for(int j=0; j<PIDX; j++) {
            int height = size_array[L*NoP*2 + j*2];
            int width = size_array[L*NoP*2 + j*2+1];
            adj_src += (unsigned long long int)(height*width*sizeof(FLOAT));
          }
              
              /* for dst, ptr */
              // adjust "dst" to tmpM[L][jj][kk]
              // adjust "ptr" to tmpIy[L][jj][kk]
          for(int i=0; i<L; i++) {
                
                /* apply error condition */
            int error_flag=0;
            for(int h=0; h<error_array_num; h++) {
              if(i==error_array[h]){
                error_flag = 1;
              }
            }
            if(error_flag != 0) {
              continue;
            }
                
            for(int j=0; j<NoC; j++) {
              for(int k=0; k<numpart[j]; k++) {
                int PIDX_tmp = PIDX_array[i*(NoC*max_numpart) + j*max_numpart + k];
                int dims0_tmp = size_array[i*NoP*2 + PIDX_tmp*2];
                int dims1_tmp = size_array[i*NoP*2 + PIDX_tmp*2+1];

                    
                adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
                adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
                    
                    
              }
            }
          }
              

          for(int i=0; i<jj; i++) {
            for(int j=0; j<numpart[i]; j++) {
              int PIDX_tmp = PIDX_array[L*(NoC*max_numpart) + i*max_numpart + j]; // PIDX_array[L][i][j]
              int dims0_tmp = size_array[L*NoP*2 + PIDX_tmp*2]; // size_array[L][PIDX_tmp*2]
              int dims1_tmp = size_array[L*NoP*2 + PIDX_tmp*2+1]; // size_array[L][PIDX_tmp*2+1]
                  
              adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
              adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
                  
            }
          }
              
          for(int j=0; j<kk; j++) {
            int PIDX_tmp = PIDX_array[L*(NoC*max_numpart) + jj*max_numpart + j]; // PIDX_array[L][jj][j]
            int dims0_tmp = size_array[L*NoP*2 + PIDX_tmp*2]; // size_array[L][PIDX_tmp*2]
            int dims1_tmp = size_array[L*NoP*2 + PIDX_tmp*2+1]; // size_array[L][PIDX_tmp*2+1]
                
            adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
            adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
          }
              
              
          FLOAT *src = (FLOAT *)adj_src;
          FLOAT *dst = (FLOAT *)adj_dst;
          int *ptr = (int *)adj_ptr;
              
          /* main calculation of di1d_x */
          XD = idx*dim0;
          dt_helper(src+XD, dst+XD, ptr+XD, step, 0, n-1, 0, n-1, a, b);
            
        }
    }
}


extern "C"
__global__
void 
dt1d_y(
  FLOAT *src_start,             // tmpM_dev
  FLOAT *dst_start,             // M_dev
  int *ptr_start,               // tmpIx_dev
  int *DID_4_array,             // DID_4_array_dev
  FLOAT *def_array,             // def_array_dev
  int NoP,                      // NoP
  int *size_array,              // pm_size_array_dev
  int *numpart,                 // numpart_jj
  int *PIDX_array,              // PIDX_array_dev
  int NoC,                      // NoC
  int max_numpart,              // max_numpart
  int interval,                 // interval
  int L_MAX,                    // L_MAX
  int *error_array,             // part_error_array_dev
  int error_array_num,           // part_error_array_num
  int pid,                       // pid
  int device_number              // device_number
       ) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int kk = blockIdx.y * blockDim.y + threadIdx.y;
  int jj = threadIdx.z;
  int L = blockIdx.z;
  int numpart_jj;
  int C_y;

  if(0<=jj && jj<NoC)
    {

      numpart_jj = numpart[jj];
      C_y = numpart_jj/device_number;

      if(numpart_jj%device_number != 0){
        C_y++;
       }
 
      kk = kk + pid * C_y;
 
      if(kk < C_y * pid  ||  kk >=  C_y * (pid + 1)){
         return ;
       }
    } else{
      return ;
    }


  if(0<=L && L<(L_MAX-interval)) 
    {
      /* loop condition */
      for(int h=0; h<error_array_num; h++) {
        if(L==error_array[h]){ 
          return;
        }
      }
      
      
      if( 0<=kk && kk<numpart_jj)
        {
          int PIDX = PIDX_array[L*(NoC*max_numpart) + jj*max_numpart + kk];
          int dim0 = size_array[L*NoP*2 + PIDX*2];

          if( idx < 0 || dim0 <= idx ) return;

          int dim1 = size_array[L*NoP*2 + PIDX*2+1];
          int step  = dim0;
          int n = dim1;
              
          int DID_4 = DID_4_array[L*(NoC*max_numpart) + jj*max_numpart + kk];
              
          FLOAT a = def_array[DID_4];   // ax
          FLOAT b = def_array[DID_4+1]; // bx
              
              /* pointer adjustment */
          unsigned long long int adj_src = (unsigned long long int)src_start;
          unsigned long long int adj_dst = (unsigned long long int)dst_start;
          unsigned long long int adj_ptr = (unsigned long long int)ptr_start;
              /* for src, dst, ptr */
              /* adjust "src" to tmpM[L][jj][kk] */
              /* adjust "dst" to M[L][jj][kk] */
              /* adjust "ptr" to tmpIx[L][jj][kk] */
          for(int i=0; i<L; i++) {

            /* apply error condition */
            int error_flag=0;
            for(int h=0; h<error_array_num; h++) {
              if(i==error_array[h]){
                error_flag = 1;
              }
            }
            if(error_flag != 0) {
              continue;
            }
                
            for(int j=0; j<NoC; j++) {
              for(int k=0; k<numpart[j]; k++) {
                    
                int PIDX_tmp = PIDX_array[i*(NoC*max_numpart) + j*max_numpart + k];
                int dims0_tmp = size_array[i*NoP*2 + PIDX_tmp*2];
                int dims1_tmp = size_array[i*NoP*2 + PIDX_tmp*2+1];
                    
                adj_src += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
                adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
                adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
                    
              }
            }
          }


          for(int i=0; i<jj; i++) {
            for(int j=0; j<numpart[i]; j++) {
              int PIDX_tmp = PIDX_array[L*(NoC*max_numpart) + i*max_numpart + j]; // PIDX_array[L][i][j]
              int dims0_tmp = size_array[L*NoP*2 + PIDX_tmp*2]; // size_array[L][PIDX_tmp*2]
              int dims1_tmp = size_array[L*NoP*2 + PIDX_tmp*2+1]; // size_array[L][PIDX_tmp*2+1]
                  
              adj_src += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
              adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
              adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
                  
            }
          }
              
          for(int j=0; j<kk; j++) {
            int PIDX_tmp = PIDX_array[L*(NoC*max_numpart) + jj*max_numpart + j];
            int dims0_tmp = size_array[L*NoP*2 + PIDX_tmp*2];
            int dims1_tmp = size_array[L*NoP*2 + PIDX_tmp*2+1];
                
            adj_src += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
            adj_dst += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(FLOAT));
            adj_ptr += (unsigned long long int)(dims0_tmp*dims1_tmp*sizeof(int));
          }
              
              
              
          FLOAT *src = (FLOAT *)adj_src;
          FLOAT *dst = (FLOAT *)adj_dst;
          int *ptr = (int *)adj_ptr;
              
 
          dt_helper(src+idx, dst+idx, ptr+idx, step, 0, n-1, 0, n-1, a, b);
          
              
        }
    }
}

/*************************************************************/
/*************************************************************/
/* original source of dt function loop */
// for (int x = 0; x < dims[1]; x++)
//   {
//     dt1d(vals+XD, tmpM+XD, tmpIy+XD, 1, dims[0], ay, by);
//     XD+=dims[0];
//   }
// for (int y = 0; y < dims[0]; y++)
//   {
//     dt1d(tmpM+y, M+y, tmpIx+y, dims[0], dims[1], ax, bx);
//   }
/*************************************************************/
/*************************************************************/



extern "C"
__global__
void
calc_a_score(
 int IWID,
 int IHEI,
 FLOAT scale,
 int padx_n,
 int pady_n,
 int *RX_array,
 int *RY_array,
 FLOAT *ac_score,
 FLOAT *score_array,
 int *ssize_array,
 int NoC,
 int *size_score_array
)
{
  int ii = blockIdx.x * blockDim.x + threadIdx.x;
  int jj = blockIdx.y * blockDim.y + threadIdx.y;

  int component_jj = threadIdx.z;

  if(0<=component_jj && component_jj < NoC) 
    {

      unsigned long long int pointer_score = (unsigned long long int)score_array;
      unsigned long long int pointer_ssize = (unsigned long long int)ssize_array;
      unsigned long long int pointer_RX = (unsigned long long int)RX_array;
      unsigned long long int pointer_RY = (unsigned long long int)RY_array;
      for(int k=0; k<component_jj; k++) {
        pointer_score += (unsigned long long int)size_score_array[k];
        pointer_ssize += (unsigned long long int)(sizeof(int));
        pointer_RX += (unsigned long long int)(sizeof(int));
        pointer_RY += (unsigned long long int)(sizeof(int));
      }

      FLOAT *score = (FLOAT *)pointer_score;
      int ssize0 = *((int *)pointer_ssize);
      int ssize1 = *((int *)pointer_ssize + sizeof(int));
      int RX = *((int *)pointer_RX);
      int RY = *((int *)pointer_RY);



      if(0<=ii && ii<IWID && 0<=jj && jj<IHEI)
        {
          int Xn = (int)((FLOAT)ii/scale+padx_n);
          int Yn = (int)((FLOAT)jj/scale+pady_n);

          
          if(Yn<ssize0 && Xn<ssize1)
            {
              FLOAT sc = score[Yn+Xn*ssize0];
              int Im_Y = jj+RY;
              int Im_X = ii+RX;
              if(Im_Y<IHEI && Im_X<IWID)
                {
                  FLOAT *PP = ac_score+Im_Y+Im_X*IHEI;
                  if(sc>*PP) *PP=sc;
                }
            }
        }
    }
  
  /*************************************************************/
  /*************************************************************/
  /* original source of calc_a_score loop */
  // for(int ii=0;ii<IWID;ii++)
  //   {
  //     int Xn=(int)((FLOAT)ii/scale+padx_n);
  
  //     for(int jj=0;jj<IHEI;jj++)
  //       {
  //         int Yn =(int)((FLOAT)jj/scale+pady_n);
  
  //         if(Yn<ssize[0] && Xn<ssize[1])
  //           {
  //             FLOAT sc = score[Yn+Xn*ssize[0]]; //get score of pixel
      
  //             int Im_Y = jj+RY;
  //             int Im_X = ii+RX;
  //             if(Im_Y<IHEI && Im_X<IWID)
  //               {
  //                 FLOAT *PP=ac_score+Im_Y+Im_X*IHEI; //consider root rectangle size
  //                 if(sc>*PP) *PP=sc;                 //save max score
  //               }
  //           }
  //       }
  //   }
  /*************************************************************/
  /*************************************************************/
  
}



#define max_i(x, y) ((x)>=(y) ? (x) : (y))
#define min_i(x, y) ((x)<=(y) ? (x) : (y))

/* atomic function dealing with double precision */
__device__ 
double 
atomicAdd_double
(
 double *address,
 double val
 )
{
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
  }while(assumed != old);
  return __longlong_as_double(old);
}  
  
extern "C"
__global__
void
calc_feature
(
 FLOAT *SRC,
 int *ISIZE, 
 FLOAT *HHist,
 int vis_R1,
 int vis_R0,
  int sbin
 )
{
  
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  const FLOAT Hcos[9]={1.0000,0.9397,0.7660,0.5000,0.1736,-0.1736,-0.5000,-0.7660,-0.9397};
  const FLOAT Hsin[9]={0.0000,0.3420,0.6428,0.8660,0.9848,0.9848,0.8660,0.6428,0.3420};


  if(1<=x && x<vis_R1 && 1<=y && y<vis_R0)
    {
      
      //input size 
      const int height=ISIZE[0]; //{268,268,134,67,233,117,203,203,177,154,89,203,154,77}
      const int width=ISIZE[1];  //{448,112,224,390,195,340,170,296,257,148,340,257,129}
      const int dims[2]={height,width};
      
      //size of Histgrams and Norm calculation space size 
      const int blocks[2] = {(int)floor(double(height)/double(sbin)+0.5),(int)floor(double(width)/double(sbin)+0.5)};//{67,112}....sbine=4
      const int BLOCK_SQ = blocks[0]*blocks[1];//{7504}...
      
      const int vp0=dims[0]-2;
      const int vp1=dims[1]-2;
      const int SQUARE =dims[0]*dims[1];
      const FLOAT SBIN = FLOAT(sbin);
      



      // for(int x=1;x<vis_R[1];x++)
      //   {
      //parameters for interpolation
      FLOAT xp=((FLOAT)x+0.5)/SBIN-0.5;
      int ixp=(int)floor(xp);
      int ixpp=ixp+1;
      int ixp_b  = ixp * blocks[0];	
      int ixpp_b = ixp_b + blocks[0];
      FLOAT vx0=xp-(FLOAT)ixp;	
      FLOAT vx1=1.0-vx0;
      bool flag1=true,flag2=true,flagX=true;
  
      if(ixp<0) 
        {
          flag1=false;
          flagX=false;
        }
      if(ixpp>=blocks[1])
        {
          flag2=false;
          flagX=false;
        }
  
      int YC=min_i(x,vp1)*dims[0];
      FLOAT *SRC_YC = SRC+YC;
  
      //      for(int y=1;y<vis_R[0];y++)
      //        {
      //first color channel
      FLOAT *s=SRC_YC+min_i(y,vp0);
      FLOAT dy=*(s+1)-*(s-1);
      FLOAT dx=*(s+dims[0])-*(s-dims[0]);
      FLOAT v=dx*dx+dy*dy;
      
      //second color channel
      s+=SQUARE;
      FLOAT dy2=*(s+1)-*(s-1);
      FLOAT dx2=*(s+dims[0])-*(s-dims[0]);
      FLOAT v2=dx2*dx2+dy2*dy2;
      
      //third color channel
      s+=SQUARE;
      FLOAT dy3=*(s+1)-*(s-1);
      FLOAT dx3=*(s+dims[0])-*(s-dims[0]);
      FLOAT v3=dx3*dx3+dy3*dy3;
      
      //pick channel with strongest gradient
      if(v2>v)
        {
          v=v2;
          dx=dx2;
          dy=dy2;
        }
      
      if(v3>v)
        {
          v=v3;
          dx=dx3;
          dy=dy3;
        }
      
      FLOAT best_dot=0.0;
      int best_o=0;
      
      //snap to one of 18 orientations
      for(int o=0;o<9;o++)
        {
          FLOAT dot=Hcos[o]*dx+Hsin[o]*dy;
          if(dot>best_dot)
            {
              best_dot=dot;
              best_o=o;
            }
          else if(-dot>best_dot)
            {
              best_dot=-dot;
              best_o=o+9;
            }
        }
      
      //Add to 4 histgrams around pixel using linear interpolation
      FLOAT yp=((FLOAT)y+0.5)/SBIN-0.5;
      int iyp=(int)floor(yp);
      int iypp=iyp+1;
      FLOAT vy0=yp-(FLOAT)iyp;
      FLOAT vy1=1.0-vy0;
      v=sqrt(v);
      int ODim=best_o*BLOCK_SQ;
      FLOAT *Htemp = HHist+ODim;
      FLOAT vx1Xv =vx1*v;
      FLOAT vx0Xv = vx0*v;
      
      if(flagX)
        {
          if(iyp>=0)
            {
              // *(Htemp+ ixp_b+iyp)+=vy1*vx1Xv; //1-少数をxyでかけたものにエッジ強度の2乗をかけたもの
              // *(Htemp+ ixpp_b+iyp)+=vy1*vx0Xv;
              if(sizeof(FLOAT) == sizeof(float)) {
                atomicAdd((float*)(Htemp + ixp_b + iyp), (float)(vy1*vx1Xv));
                atomicAdd((float*)(Htemp + ixpp_b + iyp), (float)(vy1*vx0Xv));
              }else{
                atomicAdd_double((double*)(Htemp + ixp_b + iyp), (double)(vy1*vx1Xv));
                atomicAdd_double((double*)(Htemp + ixpp_b + iyp), (double)(vy1*vx0Xv));
              }
            }
          if (iypp<blocks[0])
            {
              // *(Htemp+ ixp_b+iypp)+=vy0*vx1Xv;
              // *(Htemp+ ixpp_b+iypp)+=vy0*vx0Xv;
              if(sizeof(FLOAT) == sizeof(float)) {
                atomicAdd((float*)(Htemp + ixp_b + iypp), (float)(vy0*vx1Xv));
                atomicAdd((float*)(Htemp + ixpp_b + iypp), (float)(vy0*vx0Xv));
              }else{
                atomicAdd_double((double*)(Htemp + ixp_b + iypp), (double)(vy0*vx1Xv));
                atomicAdd_double((double*)(Htemp + ixpp_b + iypp), (double)(vy0*vx0Xv));
              }
            }
        }
      else if(flag1)
        {
          if (iyp>=0) {
             // *(Htemp+ixp_b+iyp)+=vy1*vx1Xv;
            if(sizeof(FLOAT) == sizeof(float)) {
              atomicAdd((float*)(Htemp + ixp_b + iyp), (float)(vy1*vx1Xv));
            }else{
              atomicAdd_double((double*)(Htemp + ixp_b + iyp), (double)(vy1*vx1Xv));
            }
          }
          if (iypp<blocks[0]) {
             // *(Htemp+ixp_b+iypp)+=vy0*vx1Xv;
            if(sizeof(FLOAT) == sizeof(float)) {
              atomicAdd((float*)(Htemp + ixp_b + iypp), (float)(vy0*vx1Xv));
            }else{
              atomicAdd_double((double*)(Htemp + ixp_b + iypp), (double)(vy0*vx1Xv));
            }
          }
        }
      else if(flag2)
        {
          if(iyp>=0) {
             // *(Htemp+ixpp_b+iyp)+=vy1*vx0Xv;
            if(sizeof(FLOAT)==sizeof(float)) {
              atomicAdd((float*)(Htemp + ixpp_b + iyp), (float)(vy1*vx0Xv));
            }else{
              atomicAdd_double((double*)(Htemp + ixpp_b + iyp), (double)(vy1*vx0Xv));
            }
          }
          if(iypp<blocks[0]) {
             // *(Htemp+ixpp_b+iypp)+=vy0*vx0Xv;
            if(sizeof(FLOAT)==sizeof(float)) {
              atomicAdd((float*)(Htemp + ixpp_b + iypp), (float)(vy0*vx0Xv));
            }else{
              atomicAdd_double((double*)(Htemp + ixpp_b + iypp), (double)(vy0*vx0Xv));
            }
          }
        }
      //    }
      //}




      
      
    }
  
  /*************************************************************/
  /*************************************************************/
  /* original source of calc_feature loop */
  // for(int x=1;x<vis_R[1];x++)
  //   {
  //     //parameters for interpolation
  //     FLOAT xp=((FLOAT)x+0.5)/SBIN-0.5;
  //     int ixp=(int)floor(xp);
  //     int ixpp=ixp+1;
  //     int ixp_b  = ixp * blocks[0];	
  //     int ixpp_b = ixp_b + blocks[0];
  //     FLOAT vx0=xp-(FLOAT)ixp;	
  //     FLOAT vx1=1.0-vx0;
  //     bool flag1=true,flag2=true,flagX=true;
  
  //     if(ixp<0) 
  //       {
  //         flag1=false;
  //         flagX=false;
  //       }
  //     if(ixpp>=blocks[1])
  //       {
  //         flag2=false;
  //         flagX=false;
  //       }
  
  //     int YC=min_i(x,vp1)*dims[0];
  //     FLOAT *SRC_YC = SRC+YC;
  
  //     for(int y=1;y<vis_R[0];y++)
  //       {
  //         //first color channel
  //         FLOAT *s=SRC_YC+min_i(y,vp0);
  //         FLOAT dy=*(s+1)-*(s-1);
  //         FLOAT dx=*(s+dims[0])-*(s-dims[0]);
  //         FLOAT v=dx*dx+dy*dy;
  
  //         //second color channel
  //         s+=SQUARE;
  //         FLOAT dy2=*(s+1)-*(s-1);
  //         FLOAT dx2=*(s+dims[0])-*(s-dims[0]);
  //         FLOAT v2=dx2*dx2+dy2*dy2;
  
  //         //third color channel
  //         s+=SQUARE;
  //         FLOAT dy3=*(s+1)-*(s-1);
  //         FLOAT dx3=*(s+dims[0])-*(s-dims[0]);
  //         FLOAT v3=dx3*dx3+dy3*dy3;
  
  //         //pick channel with strongest gradient
  //         if(v2>v)
  //           {
  //             v=v2;
  //             dx=dx2;
  //             dy=dy2;
  //           }
  
  //         if(v3>v)
  //           {
  //             v=v3;
  //             dx=dx3;
  //             dy=dy3;
  //           }
  
  //         FLOAT best_dot=0.0;
  //         int best_o=0;
			
  //         //snap to one of 18 orientations
  //         for(int o=0;o<9;o++)
  //           {
  //             FLOAT dot=Hcos[o]*dx+Hsin[o]*dy;
  //             if(dot>best_dot)
  //               {
  //                 best_dot=dot;
  //                 best_o=o;
  //               }
  //             else if(-dot>best_dot)
  //               {
  //                 best_dot=-dot;
  //                 best_o=o+9;
  //               }
  //           }
  
  //         //Add to 4 histgrams around pixel using linear interpolation
  //         FLOAT yp=((FLOAT)y+0.5)/SBIN-0.5;
  //         int iyp=(int)floor(yp);
  //         int iypp=iyp+1;
  //         FLOAT vy0=yp-(FLOAT)iyp;
  //         FLOAT vy1=1.0-vy0;
  //         v=sqrt(v);
  //         int ODim=best_o*BLOCK_SQ;
  //         FLOAT *Htemp = HHist+ODim;
  //         FLOAT vx1Xv =vx1*v;
  //         FLOAT vx0Xv = vx0*v;

  //         if(flagX)
  //           {
  //             if(iyp>=0)
  //               {
  //                 *(Htemp+ ixp_b+iyp)+=vy1*vx1Xv; //1-少数をxyでかけたものにエッジ強度の2乗をかけたもの
  //                 *(Htemp+ ixpp_b+iyp)+=vy1*vx0Xv;
  //               }
  //             if (iypp<blocks[0])
  //               {
  //                 *(Htemp+ ixp_b+iypp)+=vy0*vx1Xv;
  //                 *(Htemp+ ixpp_b+iypp)+=vy0*vx0Xv;
  //               }
  //           }
  //         else if(flag1)
  //           {
  //             if (iyp>=0) *(Htemp+ixp_b+iyp)+=vy1*vx1Xv;
  //             if (iypp<blocks[0]) *(Htemp+ixp_b+iypp)+=vy0*vx1Xv;
  //           }
  //         else if(flag2)
  //           {
  //             if(iyp>=0) *(Htemp+ixpp_b+iyp)+=vy1*vx0Xv;
  //             if(iypp<blocks[0]) *(Htemp+ixpp_b+iypp)+=vy0*vx0Xv;
  //           }
  //       }
  //   }
  /*************************************************************/
  /*************************************************************/
  
  
}



